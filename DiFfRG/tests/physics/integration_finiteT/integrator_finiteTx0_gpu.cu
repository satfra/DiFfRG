#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <catch2/catch_all.hpp>

#include <DiFfRG/common/math.hh>
#include <DiFfRG/common/polynomials.hh>
#include <DiFfRG/common/quadrature/quadrature_provider.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_finiteTx0_cpu.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_finiteTx0_gpu.hh>

using namespace DiFfRG;

//--------------------------------------------
// Quadrature integration

class PolyIntegrand
{
public:
  static __forceinline__ __host__ __device__ auto kernel(const double q, const double q0, const double /*k*/,
                                                         const double /*c*/, const double x0, const double x1,
                                                         const double x2, const double x3, const double x4,
                                                         const double x5, const double q0_x0, const double q0_x1,
                                                         const double q0_x2, const double q0_x3)
  {
    return (x0 + x1 * powr<1>(q) + x2 * powr<2>(q) + x3 * powr<3>(q) + x4 * powr<4>(q) + x5 * powr<5>(q)) *
           (q0_x0 + q0_x1 * powr<1>(q0) + q0_x2 * powr<2>(q0) + q0_x3 * powr<3>(q0));
  }

  static __forceinline__ __host__ __device__ auto
  constant(const double /*k*/, const double c, const double /*x0*/, const double /*x1*/, const double /*x2*/,
           const double /*x3*/, const double /*x4*/, const double /*x5*/, const double /*q0_x0*/,
           const double /*q0_x1*/, const double /*q0_x2*/, const double /*q0_x3*/)
  {
    return c;
  }
};

TEMPLATE_TEST_CASE_SIG("Test gpu momentum integrals finite T (x0)", "[integration][quadrature integration]",
                       ((int dim), dim), (2), (3), (4))
{
  const double x_extent = GENERATE(take(2, random(1., 2.)));
  const uint x0_summands = 32;
  const uint x0_int_order = 64;
  const double T = GENERATE(0., take(5, random(0.01, 1.)));
  const double k = GENERATE(take(2, random(0., 1.)));
  const double x0_extent = x0_summands * 10 * 2. * M_PI * T / k * GENERATE(take(1, random(1., 2.))) + 1000. / k;
  QuadratureProvider quadrature_provider;
  IntegratorFiniteTx0GPU<dim, double, PolyIntegrand> integrator(quadrature_provider, {{64, x0_int_order}}, x_extent,
                                                                x0_extent, x0_summands, T);

  SECTION("Volume integral")
  {
    const double q_extent = std::sqrt(x_extent * powr<2>(k));
    const double q0_extent = x0_extent * k;
    const double reference_integral = V_d(dim - 1, q_extent) / powr<dim - 1>(2. * M_PI)                 // spatial part
                                      * ((2 * x0_summands - 1) * T                                      // summands
                                         + (q0_extent - 2. * M_PI * T * x0_summands) * 2. / (2. * M_PI) // integral
                                        );

    const double integral = integrator.request(k, 0., 1., 0., 0., 0., 0., 0., 1., 0., 0., 0.).get();

    if (!is_close(reference_integral, integral, dim == 2 ? 1e-2 : 5e-5)) {
      std::cerr << "dim: " << dim << "| reference: " << reference_integral << "| integral: " << integral
                << "| relative error: " << std::abs(reference_integral - integral) / std::abs(reference_integral)
                << std::endl;
    }
    CHECK(isfinite(integral));
    CHECK(is_close(reference_integral, integral, dim == 2 ? 1e-2 : 5e-5));
  }

  SECTION("Random polynomials : check consistency with CPU case.")
  {
    IntegratorFiniteTx0TBB<dim, double, PolyIntegrand> integrator_cpu(quadrature_provider, {{64, 12}}, x_extent,
                                                                      x0_extent, x0_summands, T);

    constexpr uint take_n = 2;

    const auto poly = Polynomial({
        dim == 2 ? 0. : GENERATE(take(take_n, random(-1., 1.))), // x0
        GENERATE(take(take_n, random(-1., 1.))),                 // x1
        GENERATE(take(1, random(-1., 1.))),                      // x2
        GENERATE(take(1, random(-1., 1.))),                      // x3
        GENERATE(take(1, random(-1., 1.))),                      // x4
        GENERATE(take(1, random(-1., 1.)))                       // x5
    });
    const auto q0_poly = Polynomial({
        GENERATE(take(take_n, random(-1., 1.))), // x0
        GENERATE(take(1, random(-1., 1.))),      // x1
        GENERATE(take(1, random(-1., 1.))),      // x2
        GENERATE(take(1, random(-1., 1.)))       // x3
    });

    const double k = GENERATE(take(take_n, random(0., 1.)));
    const double q_extent = std::sqrt(x_extent * powr<2>(k));
    const double q0_extent = x0_extent * k;
    const double constant = GENERATE(take(take_n, random(-1., 1.)));

    const double int_gpu = integrator.get(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], q0_poly[0],
                                          q0_poly[1], q0_poly[2], q0_poly[3]);
    const double int_cpu = integrator_cpu.get(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5],
                                              q0_poly[0], q0_poly[1], q0_poly[2], q0_poly[3]);

    if (!is_close(int_gpu, int_cpu, 1e-8)) {
      std::cerr << "dim: " << dim << "| GPU: " << int_gpu << "| CPU: " << int_cpu
                << "| relative error: " << std::abs(int_gpu - int_cpu) / std::abs(int_gpu) << std::endl;
    }
    CHECK(isfinite(int_cpu));
    CHECK(isfinite(int_gpu));
    CHECK(is_close(int_gpu, int_cpu, 1e-8));
  }
}