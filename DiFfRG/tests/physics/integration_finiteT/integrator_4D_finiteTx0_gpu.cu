#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <catch2/catch_all.hpp>

#include <DiFfRG/common/math.hh>
#include <DiFfRG/common/polynomials.hh>
#include <DiFfRG/common/quadrature/quadrature_provider.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_4D_finiteTx0_cpu.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_4D_finiteTx0_gpu.hh>

using namespace DiFfRG;

//--------------------------------------------
// Quadrature integration

class PolyIntegrand
{
public:
  static __forceinline__ __host__ __device__ auto
  kernel(const double q, const double cos, const double phi, const double q0, const double /*k*/, const double /*c*/,
         const double x0, const double x1, const double x2, const double x3, const double x4, const double x5,
         const double cos_x0, const double cos_x1, const double cos_x2, const double cos_x3, const double phi_x0,
         const double phi_x1, const double phi_x2, const double phi_x3, const double q0_x0, const double q0_x1,
         const double q0_x2, const double q0_x3)
  {
    return (x0 + x1 * powr<1>(q) + x2 * powr<2>(q) + x3 * powr<3>(q) + x4 * powr<4>(q) + x5 * powr<5>(q)) *
           (cos_x0 + cos_x1 * powr<1>(cos) + cos_x2 * powr<2>(cos) + cos_x3 * powr<3>(cos)) *
           (phi_x0 + phi_x1 * powr<1>(phi) + phi_x2 * powr<2>(phi) + phi_x3 * powr<3>(phi)) *
           (q0_x0 + q0_x1 * powr<1>(q0) + q0_x2 * powr<2>(q0) + q0_x3 * powr<3>(q0));
  }

  static __forceinline__ __host__ __device__ auto
  constant(const double /*k*/, const double c, const double /*x0*/, const double /*x1*/, const double /*x2*/,
           const double /*x3*/, const double /*x4*/, const double /*x5*/, const double /*cos_x0*/,
           const double /*cos_x1*/, const double /*cos_x2*/, const double /*cos_x3*/, const double /*phi_x0*/,
           const double /*phi_x1*/, const double /*phi_x2*/, const double /*phi_x3*/, const double /*q0_x0*/,
           const double /*q0_x1*/, const double /*q0_x2*/, const double /*q0_x3*/)
  {
    return c;
  }
};

TEST_CASE("Test 4D gpu momentum integrals with finite T (x0)", "[4D integration][quadrature integration]")
{
  constexpr int dim = 4;

  const double x_extent = GENERATE(take(2, random(1., 2.)));
  const double x0_summands = 8;
  const double T = GENERATE(0., take(3, random(0.01, 1.)));
  const double k = GENERATE(take(2, random(0., 1.)));
  const double x0_extent = x0_summands * 10 * 2. * M_PI * T / k * GENERATE(take(1, random(1., 2.))) + 1000. / k;
  QuadratureProvider quadrature_provider;
  Integrator4DFiniteTx0GPU<double, PolyIntegrand> integrator(quadrature_provider, {{64, 12, 12, 12}}, x_extent,
                                                             x0_extent, x0_summands, T);

  SECTION("Volume integral")
  {
    const double q_extent = std::sqrt(x_extent * powr<2>(k));
    const double q0_extent = x0_extent * k;
    const double reference_integral = V_d(dim - 1, q_extent) / powr<dim - 1>(2. * M_PI)                 // spatial part
                                      * ((2 * x0_summands - 1) * T                                      // summands
                                         + (q0_extent - 2. * M_PI * T * x0_summands) * 2. / (2. * M_PI) // integral
                                        );

    const double integral =
        integrator.request(k, 0., 1., 0., 0., 0., 0., 0., 1., 0., 0., 0., 1., 0., 0., 0., 1., 0., 0., 0.).get();

    if (!is_close(reference_integral, integral, dim == 2 ? 1e-2 : 5e-5)) {
      std::cerr << "dim: " << dim << "| reference: " << reference_integral << "| integral: " << integral
                << "| relative error: " << std::abs(reference_integral - integral) / std::abs(reference_integral)
                << std::endl;
    }
    CHECK(isfinite(integral));
    CHECK(is_close(reference_integral, integral, dim == 2 ? 1e-2 : 5e-5));
  }
}