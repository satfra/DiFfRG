#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <catch2/catch_all.hpp>

#include <DiFfRG/common/math.hh>
#include <DiFfRG/common/polynomials.hh>
#include <DiFfRG/common/quadrature/quadrature_provider.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_4D_finiteTx0_cpu.hh>
#include <DiFfRG/physics/integration_finiteT/integrator_4D_finiteTx0_gpu.hh>

using namespace DiFfRG;

//--------------------------------------------
// Quadrature integration

class PolyIntegrand
{
public:
  static __forceinline__ __host__ __device__ auto
  kernel(const double q, const double cos, const double phi, const double q0, const double /*k*/, const double /*c*/,
         const double x0, const double x1, const double x2, const double x3, const double x4, const double x5,
         const double cos_x0, const double cos_x1, const double cos_x2, const double cos_x3, const double phi_x0,
         const double phi_x1, const double phi_x2, const double phi_x3, const double q0_x0, const double q0_x1,
         const double q0_x2, const double q0_x3)
  {
    return (x0 + x1 * powr<1>(q) + x2 * powr<2>(q) + x3 * powr<3>(q) + x4 * powr<4>(q) + x5 * powr<5>(q)) *
           (cos_x0 + cos_x1 * powr<1>(cos) + cos_x2 * powr<2>(cos) + cos_x3 * powr<3>(cos)) *
           (phi_x0 + phi_x1 * powr<1>(phi) + phi_x2 * powr<2>(phi) + phi_x3 * powr<3>(phi)) *
           (q0_x0 + q0_x1 * powr<1>(q0) + q0_x2 * powr<2>(q0) + q0_x3 * powr<3>(q0));
  }

  static __forceinline__ __host__ __device__ auto
  constant(const double /*k*/, const double c, const double /*x0*/, const double /*x1*/, const double /*x2*/,
           const double /*x3*/, const double /*x4*/, const double /*x5*/, const double /*cos_x0*/,
           const double /*cos_x1*/, const double /*cos_x2*/, const double /*cos_x3*/, const double /*phi_x0*/,
           const double /*phi_x1*/, const double /*phi_x2*/, const double /*phi_x3*/, const double /*q0_x0*/,
           const double /*q0_x1*/, const double /*q0_x2*/, const double /*q0_x3*/)
  {
    return c;
  }
};

TEST_CASE("Benchmark 4D momentum integrals", "[4D integration][quadrature integration]")
{
  constexpr int dim = 4;

  const double x_extent = GENERATE(take(1, random(1., 2.)));
  QuadratureProvider quadrature_provider;

  constexpr uint take_n = 1;
  const auto poly = Polynomial({
      dim == 1 ? 0. : GENERATE(take(take_n, random(-1., 1.))), // x0
      GENERATE(take(take_n, random(-1., 1.))),                 // x1
      GENERATE(take(take_n, random(-1., 1.))),                 // x2
      GENERATE(take(1, random(-1., 1.))),                      // x3
      GENERATE(take(1, random(-1., 1.))),                      // x4
      GENERATE(take(1, random(-1., 1.))),                      // x5
  });
  const auto cos_poly = Polynomial({
      GENERATE(take(take_n, random(-1., 1.))), // x0
      GENERATE(take(1, random(-1., 1.))),      // x1
      GENERATE(take(1, random(-1., 1.))),      // x2
      GENERATE(take(1, random(-1., 1.)))       // x3
  });
  const auto phi_poly = Polynomial({
      GENERATE(take(take_n, random(-1., 1.))), // x0
      GENERATE(take(1, random(-1., 1.))),      // x1
      GENERATE(take(1, random(-1., 1.))),      // x2
      GENERATE(take(1, random(-1., 1.)))       // x3
  });
  const auto x0_poly = Polynomial({
      GENERATE(take(take_n, random(-1., 1.))), // x0
      GENERATE(take(1, random(-1., 1.))),      // x1
      GENERATE(take(1, random(-1., 1.))),      // x2
      GENERATE(take(1, random(-1., 1.)))       // x3
  });

  const double k = GENERATE(take(take_n, random(0., 1.)));
  const double q_extent = std::sqrt(x_extent * powr<2>(k));
  const double constant = GENERATE(take(take_n, random(-1., 1.)));
  const double x0_summands = 8;
  const double T = GENERATE(take(take_n, random(0.5, 1.)));
  const double x0_extent = x0_summands * 10 * 2. * M_PI * T / k * GENERATE(take(1, random(1., 2.)));

  auto int_poly = poly;
  std::vector<double> coeff_integrand(dim, 0.);
  coeff_integrand[dim - 1] = 1.;
  int_poly *= Polynomial(coeff_integrand);
  {
    Integrator4DFiniteTx0GPU<double, PolyIntegrand> integrator(quadrature_provider, {{32, 8, 8, 8}}, x_extent,
                                                               x0_extent, x0_summands, T);
    BENCHMARK_ADVANCED("GPU")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        const double integral =
            integrator
                .request(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], cos_poly[0], cos_poly[1],
                         cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1], phi_poly[2], phi_poly[3], x0_poly[0],
                         x0_poly[1], x0_poly[2], x0_poly[3])
                .get();
      });
    };
    BENCHMARK_ADVANCED("GPU 128x")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        std::vector<std::future<double>> futures;
        for (int i = 0; i < 128; ++i)
          futures.emplace_back(
              std::move(integrator.request(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5],
                                           cos_poly[0], cos_poly[1], cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1],
                                           phi_poly[2], phi_poly[3], x0_poly[0], x0_poly[1], x0_poly[2], x0_poly[3])));
        for (auto &f : futures)
          f.get();
      });
    };
    BENCHMARK_ADVANCED("get GPU 128x")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        for (int i = 0; i < 128; ++i)
          integrator.get(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], cos_poly[0], cos_poly[1],
                         cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1], phi_poly[2], phi_poly[3], x0_poly[0],
                         x0_poly[1], x0_poly[2], x0_poly[3]);
      });
    };
  }
  {
    Integrator4DFiniteTx0TBB<double, PolyIntegrand> integrator(quadrature_provider, {{32, 8, 8, 8}}, x_extent,
                                                               x0_extent, x0_summands, T);
    BENCHMARK_ADVANCED("CPU")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        const double integral =
            integrator
                .request(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], cos_poly[0], cos_poly[1],
                         cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1], phi_poly[2], phi_poly[3], x0_poly[0],
                         x0_poly[1], x0_poly[2], x0_poly[3])
                .get();
      });
    };
    BENCHMARK_ADVANCED("get CPU")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        const double integral =
            integrator.get(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], cos_poly[0], cos_poly[1],
                           cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1], phi_poly[2], phi_poly[3], x0_poly[0],
                           x0_poly[1], x0_poly[2], x0_poly[3]);
      });
    };
    BENCHMARK_ADVANCED("CPU 128x")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        std::vector<std::future<double>> futures;
        for (int i = 0; i < 128; ++i)
          futures.emplace_back(
              std::move(integrator.request(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5],
                                           cos_poly[0], cos_poly[1], cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1],
                                           phi_poly[2], phi_poly[3], x0_poly[0], x0_poly[1], x0_poly[2], x0_poly[3])));
        for (auto &f : futures)
          f.get();
      });
    };
    BENCHMARK_ADVANCED("get CPU 128x")(Catch::Benchmark::Chronometer meter)
    {
      meter.measure([&] {
        for (int i = 0; i < 128; ++i)
          integrator.get(k, constant, poly[0], poly[1], poly[2], poly[3], poly[4], poly[5], cos_poly[0], cos_poly[1],
                         cos_poly[2], cos_poly[3], phi_poly[0], phi_poly[1], phi_poly[2], phi_poly[3], x0_poly[0],
                         x0_poly[1], x0_poly[2], x0_poly[3]);
      });
    };
  }
}