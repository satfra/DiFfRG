// DiFfRG
#include <DiFfRG/common/quadrature/quadrature_provider.hh>

namespace DiFfRG
{
  QuadratureProvider::QuadratureProvider() {}
  QuadratureProvider::~QuadratureProvider() = default;

  const std::vector<double> &QuadratureProvider::get_points_d(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].nodes();
  }
  const std::vector<double> &QuadratureProvider::get_weights_d(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].weights();
  }

  const std::vector<float> &QuadratureProvider::get_points_f(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].nodes();
  }
  const std::vector<float> &QuadratureProvider::get_weights_f(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].weights();
  }

  const std::vector<double> &QuadratureProvider::get_matsubara_points_d(const double T, const double typical_E)
  {
    const auto size = MatsubaraQuadrature<double>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_d.find(size) == matsubara_quadrature_d.end()) compute_matsubara_quadrature_d(T, typical_E);
    return matsubara_quadrature_d[size].nodes();
  }
  const std::vector<double> &QuadratureProvider::get_matsubara_weights_d(const double T, const double typical_E)
  {
    const auto size = MatsubaraQuadrature<double>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_d.find(size) == matsubara_quadrature_d.end()) compute_matsubara_quadrature_d(T, typical_E);
    return matsubara_quadrature_d[size].weights();
  }

  const std::vector<float> &QuadratureProvider::get_matsubara_points_f(const float T, const float typical_E)
  {
    const auto size = MatsubaraQuadrature<float>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_f.find(size) == matsubara_quadrature_f.end()) compute_matsubara_quadrature_f(T, typical_E);
    return matsubara_quadrature_f[size].nodes();
  }
  const std::vector<float> &QuadratureProvider::get_matsubara_weights_f(const float T, const float typical_E)
  {
    const auto size = MatsubaraQuadrature<float>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_f.find(size) == matsubara_quadrature_f.end()) compute_matsubara_quadrature_f(T, typical_E);
    return matsubara_quadrature_f[size].weights();
  }

#ifdef __HIPCC__
  const double *QuadratureProvider::get_device_points_d(const uint quadrature_size, const int device,
                                                        const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].device_nodes();
  }
  const double *QuadratureProvider::get_device_weights_d(const uint quadrature_size, const int device,
                                                         const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].device_weights();
  }

  const float *QuadratureProvider::get_device_points_f(const uint quadrature_size, const int device,
                                                       const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].device_nodes();
  }
  const float *QuadratureProvider::get_device_weights_f(const uint quadrature_size, const int device,
                                                        const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].device_weights();
  }

  const double *QuadratureProvider::get_device_matsubara_points_d(const double T, const double typical_E,
                                                                  const int device)
  {
    const auto size = MatsubaraQuadrature<double>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_d.find(size) == matsubara_quadrature_d.end()) compute_matsubara_quadrature_d(T, typical_E);
    return matsubara_quadrature_d[size].device_nodes();
  }
  const double *QuadratureProvider::get_device_matsubara_weights_d(const double T, const double typical_E,
                                                                   const int device)
  {
    const auto size = MatsubaraQuadrature<double>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_d.find(size) == matsubara_quadrature_d.end()) compute_matsubara_quadrature_d(T, typical_E);
    return matsubara_quadrature_d[size].device_weights();
  }

  const float *QuadratureProvider::get_device_matsubara_points_f(const float T, const float typical_E, const int device)
  {
    const auto size = MatsubaraQuadrature<float>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_f.find(size) == matsubara_quadrature_f.end()) compute_matsubara_quadrature_f(T, typical_E);
    return matsubara_quadrature_f[size].device_nodes();
  }
  const float *QuadratureProvider::get_device_matsubara_weights_f(const float T, const float typical_E,
                                                                  const int device)
  {
    const auto size = MatsubaraQuadrature<float>::predict_size(T, typical_E);
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (matsubara_quadrature_f.find(size) == matsubara_quadrature_f.end()) compute_matsubara_quadrature_f(T, typical_E);
    return matsubara_quadrature_f[size].device_weights();
  }
#endif

  void QuadratureProvider::compute_quadrature_d(const uint quadrature_size, const QuadratureType _t)
  {
    auto [it, success] = quadrature_d[(size_t)_t].insert(std::make_pair(quadrature_size, Quadrature<double>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(quadrature_size, _t);
  }

  void QuadratureProvider::compute_quadrature_f(const uint quadrature_size, const QuadratureType _t)
  {
    auto [it, success] = quadrature_f[(size_t)_t].insert(std::make_pair(quadrature_size, Quadrature<float>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(quadrature_size, _t);
  }

  void QuadratureProvider::compute_matsubara_quadrature_d(const double T, const double typical_E)
  {
    const auto size = MatsubaraQuadrature<double>::predict_size(T, typical_E);
    auto [it, success] = matsubara_quadrature_d.insert(std::make_pair(size, MatsubaraQuadrature<double>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(T, typical_E);
  }

  void QuadratureProvider::compute_matsubara_quadrature_f(const float T, const float typical_E)
  {
    const auto size = MatsubaraQuadrature<float>::predict_size(T, typical_E);
    auto [it, success] = matsubara_quadrature_f.insert(std::make_pair(size, MatsubaraQuadrature<float>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(T, typical_E);
  }
} // namespace DiFfRG