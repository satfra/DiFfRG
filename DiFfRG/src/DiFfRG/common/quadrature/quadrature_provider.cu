// DiFfRG
#include <DiFfRG/common/quadrature/quadrature_provider.hh>

namespace DiFfRG
{
  QuadratureProvider::QuadratureProvider() {}
  QuadratureProvider::~QuadratureProvider() = default;

  const std::vector<double> &QuadratureProvider::get_points_d(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].nodes();
  }
  const std::vector<double> &QuadratureProvider::get_weights_d(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].weights();
  }

  const std::vector<float> &QuadratureProvider::get_points_f(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].nodes();
  }
  const std::vector<float> &QuadratureProvider::get_weights_f(const uint quadrature_size, const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].weights();
  }

#ifdef __HIPCC__
  const double *QuadratureProvider::get_device_points_d(const uint quadrature_size, const int device,
                                                        const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].device_nodes();
  }
  const double *QuadratureProvider::get_device_weights_d(const uint quadrature_size, const int device,
                                                         const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_d[(size_t)_t].find(quadrature_size) == quadrature_d[(size_t)_t].end())
      compute_quadrature_d(quadrature_size, _t);
    return quadrature_d[(size_t)_t][quadrature_size].device_weights();
  }

  const float *QuadratureProvider::get_device_points_f(const uint quadrature_size, const int device,
                                                       const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].device_nodes();
  }
  const float *QuadratureProvider::get_device_weights_f(const uint quadrature_size, const int device,
                                                        const QuadratureType _t)
  {
    // if the quadrature of size quadrature_size is not yet computed, compute it
    if (quadrature_f[(size_t)_t].find(quadrature_size) == quadrature_f[(size_t)_t].end())
      compute_quadrature_f(quadrature_size, _t);
    return quadrature_f[(size_t)_t][quadrature_size].device_weights();
  }
#endif

  void QuadratureProvider::compute_quadrature_d(const uint quadrature_size, const QuadratureType _t)
  {
    auto [it, success] = quadrature_d[(size_t)_t].insert(std::make_pair(quadrature_size, Quadrature<double>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(quadrature_size, _t);
  }

  void QuadratureProvider::compute_quadrature_f(const uint quadrature_size, const QuadratureType _t)
  {
    auto [it, success] = quadrature_f[(size_t)_t].insert(std::make_pair(quadrature_size, Quadrature<float>()));
    if (!success) throw std::runtime_error("Failed to insert quadrature into map");
    it->second.reinit(quadrature_size, _t);
  }
} // namespace DiFfRG