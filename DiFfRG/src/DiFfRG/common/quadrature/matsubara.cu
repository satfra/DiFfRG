#include "hip/hip_runtime.h"
// DiFfRG
#include <DiFfRG/common/cuda_prefix.hh>
#include <DiFfRG/common/math.hh>
#include <DiFfRG/common/quadrature/matsubara.hh>
#include <DiFfRG/common/quadrature/quadrature.hh>

namespace DiFfRG
{
  template <typename NT> size_t MatsubaraQuadrature<NT>::predict_size(const NT T, const NT typical_E, const size_t step)
  {
    const NT E_max = 10 * std::abs(typical_E);
    size_t size = 5 + int(std::sqrt(4 * E_max / (M_PI * M_PI * std::abs(T))));
    size = (size_t)std::ceil(size / (double)step) * step;
    return size;
  }

  template <typename NT>
  MatsubaraQuadrature<NT>::MatsubaraQuadrature(const NT T, const NT typical_E, const size_t step, const size_t min_size,
                                               const size_t max_size)
  {
    reinit(T, typical_E, step, min_size, max_size);
  }

  template <typename NT> MatsubaraQuadrature<NT>::MatsubaraQuadrature() {}

  template <typename NT>
  void MatsubaraQuadrature<NT>::reinit(const NT T, const NT typical_E, const size_t step, const size_t min_size,
                                       const size_t max_size)
  {
    this->T = T;
    this->typical_E = typical_E;

#ifdef USE_CUDA
    device_x.clear();
    device_w.clear();
#endif

    // Determine the number of nodes in the quadrature rule.
    m_size = predict_size(T, typical_E, step);
    m_size = std::max(min_size, std::min(max_size, m_size));

    // construct the recurrence relation for the quadrature rule from [1]
    std::vector<NT> a(m_size, 0.);
    std::vector<NT> b(m_size, 0.);

    for (size_t j = 0; j < m_size; ++j) {
      const double j1 = j + 1;
      a[j] = 2 * powr<2>(M_PI) / (4 * j + 1) / (4 * j + 5);
      b[j] = powr<4>(M_PI) / ((4 * j1 - 1) * (4 * j1 + 3)) / powr<2>(4 * j1 + 1);
    }
    a[0] = powr<2>(M_PI) / 15.;

    const NT mu0 = powr<2>(M_PI) / 6.;

    // compute the nodes and weights of the quadrature rule
    make_quadrature(a, b, mu0, x, w);

    // normalize the weights and scale the nodes
    for (size_t i = 0; i < m_size; ++i) {
      w[i] = T * w[i] / x[i];
      x[i] = 2. * M_PI * T / std::sqrt(x[i]);
    }
  }

  template <typename NT> const std::vector<NT> &MatsubaraQuadrature<NT>::nodes() const { return x; }
  template <typename NT> const std::vector<NT> &MatsubaraQuadrature<NT>::weights() const { return w; }

  template <typename NT> size_t MatsubaraQuadrature<NT>::size() const { return m_size; }
  template <typename NT> NT MatsubaraQuadrature<NT>::get_T() const { return T; }
  template <typename NT> NT MatsubaraQuadrature<NT>::get_typical_E() const { return typical_E; }

#ifdef USE_CUDA
  template <typename NT> const NT *MatsubaraQuadrature<NT>::device_nodes()
  {
    move_device_data();
#ifdef __HIPCC__
    return thrust::raw_pointer_cast(device_x.data());
#else
    return device_x.data().get();
#endif
  }

  template <typename NT> const NT *MatsubaraQuadrature<NT>::device_weights()
  {
    move_device_data();
#ifdef __HIPCC__
    return thrust::raw_pointer_cast(device_w.data());
#else
    return device_w.data().get();
#endif
  }
#endif

  template <typename NT> void MatsubaraQuadrature<NT>::move_device_data()
  {
#ifdef __HIPCC__
    if (device_x.size() == 0) {
      device_x.resize(m_size);
      device_w.resize(m_size);
      thrust::copy(x.begin(), x.end(), device_x.begin());
      thrust::copy(w.begin(), w.end(), device_w.begin());
    }
#endif
  }

  // explicit instantiation
  template class MatsubaraQuadrature<double>;
  template class MatsubaraQuadrature<float>;
} // namespace DiFfRG